#include "hip/hip_runtime.h"
#include <cuda/barrier>
#include <hip/hip_cooperative_groups.h>

__device__ void compute(float *data, int curr_iteration);

__global__ void split_arrive_wait(int iteration_count, float *data) {
    using barrier = cuda::barrier<cuda::thread_scope_block>;
    __shared__ barrier bar;
    auto block = cooperative_groups::this_thread_block();
    if (block.thread_rank() == 0) {
        init(&bar, block.size()); // Initialize the barrier with expected arrival count
    }
    block.sync();
    for (int curr_iter = 0; curr_iter < iteration_count; ++curr_iter) {
        /* code before arrive */
        barrier::arrival_token token = bar.arrive(); /* this thread arrives. Arrival￿ does not block a thread */
        compute(data, curr_iter);
        bar.wait(std::move(token)); /* wait for all threads participating in the barrier￿ to complete bar.arrive()*/
        /* code after wait */
    }
}

#include <cuda/barrier>
#include <hip/hip_cooperative_groups.h>

using barrier = cuda::barrier<cuda::thread_scope_block>;

__device__ void producer(barrier ready[], barrier filled[], float* buffer, float* in, int N, int buffer_len)
{
    for (int i = 0; i < (N/buffer_len); ++i) {
        ready[i%2].arrive_and_wait(); /* wait for buffer_(i%2) to be ready to be filled */
        /* produce, i.e., fill in, buffer_(i%2)  */
        barrier::arrival_token token = filled[i%2].arrive(); /* buffer_(i%2) is filled */
    }
}

__device__ void consumer(barrier ready[], barrier filled[], float* buffer, float* out, int N, int buffer_len)
{
    barrier::arrival_token token1 = ready[0].arrive(); /* buffer_0 is ready for initial fill */
    barrier::arrival_token token2 = ready[1].arrive(); /* buffer_1 is ready for initial fill */
    for (int i = 0; i < (N/buffer_len); ++i) {
        filled[i%2].arrive_and_wait(); /* wait for buffer_(i%2) to be filled */
        /* consume buffer_(i%2) */
        barrier::arrival_token token = ready[i%2].arrive(); /* buffer_(i%2) is ready to be re-filled */
    }
}

//N is the total number of float elements in arrays in and out
__global__ void producer_consumer_pattern(int N, int buffer_len, float* in, float* out) {

    // Shared memory buffer declared below is of size 2 * buffer_len
    // so that we can alternatively work between two buffers.
    // buffer_0 = buffer and buffer_1 = buffer + buffer_len
    __shared__ extern float buffer[];

    // bar[0] and bar[1] track if buffers buffer_0 and buffer_1 are ready to be filled,
    // while bar[2] and bar[3] track if buffers buffer_0 and buffer_1 are filled-in respectively
    __shared__ barrier bar[4];


    auto block = cooperative_groups::this_thread_block();
    if (block.thread_rank() < 4)
        init(bar + block.thread_rank(), block.size());
    block.sync();

    if (block.thread_rank() < warpSize)
        producer(bar, bar+2, buffer, in, N, buffer_len);
    else
        consumer(bar, bar+2, buffer, out, N, buffer_len);
}