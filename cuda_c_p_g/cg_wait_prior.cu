#include "hip/hip_runtime.h"
/// This example streams elementsPerThreadBlock worth of data from global memory
/// into a limited sized shared memory (elementsInShared) block to operate on in
/// multiple (two) stages. As stage N is kicked off, we can wait on and operate on stage N-1.
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>

namespace cg = cooperative_groups;

__global__ void kernel(int* global_data) {
    cg::thread_block tb = cg::this_thread_block();
    const size_t elementsPerThreadBlock = 16 * 1024 + 64;
    const size_t elementsInShared = 128;
    __align__(16) __shared__ int local_smem[2][elementsInShared];
    int stage = 0;
    // First kick off an extra request
    size_t copy_count = elementsInShared;
    size_t index = copy_count;
    cg::memcpy_async(tb, local_smem[stage], elementsInShared, global_data, elementsPerThreadBlock - index);
    while (index < elementsPerThreadBlock) {
        // Now we kick off the next request...
        cg::memcpy_async(tb, local_smem[stage ^ 1], elementsInShared, global_data + index, elementsPerThreadBlock - index);
        // ... but we wait on the one before it
        cg::wait_prior<1>(tb);

        // Its now available and we can work with local_smem[stage] here
        // (...)
        //

        // Calculate the amount fo data that was actually copied, for the next iteration.
        copy_count = min(elementsInShared, elementsPerThreadBlock - index);
        index += copy_count;

        // A cg::sync(tb) might be needed here depending on whether
        // the work done with local_smem[stage] can release threads to race ahead or not
        // Wrap to the next stage
        stage ^= 1;
    }
    cg::wait(tb);
    // The last local_smem[stage] can be handled here
}