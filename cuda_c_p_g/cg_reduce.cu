#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

/// Calculate approximate standard deviation of integers in vec
__device__ int std_dev(const cg::thread_block_tile<32>& tile, int *vec, int length) {
    int thread_sum = 0;

    // calculate average first
    for (int i = tile.thread_rank(); i < length; i += tile.num_threads()) {
        thread_sum += vec[i];
    }
    // cg::plus<int> allows cg::reduce() to know it can use hardware acceleration for addition
    int avg = cg::reduce(tile, thread_sum, cg::plus<int>()) / length;

    int thread_diffs_sum = 0;
    for (int i = tile.thread_rank(); i < length; i += tile.num_threads()) {
        int diff = vec[i] - avg;
        thread_diffs_sum += diff * diff;
    }

    // temporarily use floats to calculate the square root
    float diff_sum = static_cast<float>(cg::reduce(tile, thread_diffs_sum, cg::plus<int>())) / length;

    return static_cast<int>(sqrtf(diff_sum));
}

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg=cooperative_groups;

/// The following example accepts input in *A and outputs a result into *sum
/// It spreads the data equally within the block
__device__ void block_reduce(const int* A, int count, cuda::atomic<int, cuda::thread_scope_block>& total_sum) {
    auto block = cg::this_thread_block();
    auto tile = cg::tiled_partition<32>(block);
    int thread_sum = 0;

    // Stride loop over all values, each thread accumulates its part of the array.
    for (int i = block.thread_rank(); i < count; i += block.size()) {
        thread_sum += A[i];
    }

    // reduce thread sums across the tile, add the result to the atomic
    // cg::plus<int> allows cg::reduce() to know it can use hardware acceleration for addition
    cg::reduce_update_async(tile, total_sum, thread_sum, cg::plus<int>());

    // synchronize the block, to ensure all async reductions are ready
    block.sync();
}