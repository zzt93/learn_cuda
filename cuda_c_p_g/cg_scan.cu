#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/scan.h>
namespace cg = cooperative_groups;

// put data from input into output only if it passes test_fn predicate
template<typename Group, typename Data, typename TyFn>
__device__ int stream_compaction(Group &g, Data *input, int count, TyFn&& test_fn, Data *output) {
    int per_thread = count / g.num_threads();
    int thread_start = min(g.thread_rank() * per_thread, count);
    int my_count = min(per_thread, count - thread_start);

    // get all passing items from my part of the input
    //  into a contagious part of the array and count them.
    int i = thread_start;
    while (i < my_count + thread_start) {
        if (test_fn(input[i])) {
            i++;
        }
        else {
            my_count--;
            input[i] = input[my_count + thread_start];
        }
    }

    // scan over counts from each thread to calculate my starting
    //  index in the output
    int my_idx = cg::exclusive_scan(g, my_count);

    for (i = 0; i < my_count; ++i) {
        output[my_idx + i] = input[thread_start + i];
    }
    // return the total number of items in the output
    return g.shfl(my_idx + my_count, g.num_threads() - 1);
}

#include <stdio.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/scan.h>
namespace cg = cooperative_groups;

__global__ void kernel() {
    auto thread_block = cg::this_thread_block();
    auto tile = cg::tiled_partition<8>(thread_block);
    unsigned int val = cg::inclusive_scan(tile, tile.thread_rank());
    printf("%u: %u\n", tile.thread_rank(), val);
}

/*  prints for each group:
    0: 0
    1: 1
    2: 3
    3: 6
    4: 10
    5: 15
    6: 21
    7: 28
*/

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/scan.h>
namespace cg = cooperative_groups;

// Buffer partitioning is static to make the example easier to follow,
// but any arbitrary dynamic allocation scheme can be implemented by replacing this function.
__device__ int calculate_buffer_space_needed(cg::thread_block_tile<32>& tile) {
    return tile.thread_rank() % 2 + 1;
}

__device__ int my_thread_data(int i) {
    return i;
}

__global__ void kernel() {
    __shared__ extern int buffer[];
    __shared__ cuda::atomic<int, cuda::thread_scope_block> buffer_used;

    auto block = cg::this_thread_block();
    auto tile = cg::tiled_partition<32>(block);
    buffer_used = 0;
    block.sync();

    // each thread calculates buffer size it needs
    int buf_needed = calculate_buffer_space_needed(tile);

    // scan over the needs of each thread, result for each thread is an offset
    // of that thread’s part of the buffer. buffer_used is atomically updated with
    // the sum of all thread's inputs, to correctly offset other tile’s allocations
    int buf_offset =
            cg::exclusive_scan_update(tile, buffer_used, buf_needed);

    // each thread fills its own part of the buffer with thread specific data
    for (int i = 0 ; i < buf_needed ; ++i) {
        buffer[buf_offset + i] = my_thread_data(i);
    }

    block.sync();
    // buffer_used now holds total amount of memory allocated
    // buffer is {0, 0, 1, 0, 0, 1 ...};

}