#include "hip/hip_runtime.h"
//
// Created by ByteDance on 2025/6/3.
//
//%%writefile cnn.cu

#include <hip/hip_runtime.h>
#include <stdio.h>


#define CHECK(call)\
{\
  const hipError_t error=call;\
  if(error!=hipSuccess)\
  {\
      printf("ERROR: %s:%d,",__FILE__,__LINE__);\
      printf("code:%d,reason:%s\n",error,hipGetErrorString(error));\
      exit(1);\
  }\
}

__global__ void convolve2d(float *img, int img_nx, int img_ny, float *kernel, int k_nx, int k_ny, float *out, int out_nx, int out_ny, int stride, int padding, int padding_value) {
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;
    if (ix >= out_nx || iy >= out_ny) return;

    int img_x_s = -padding + ix * stride;
    int img_y_s = -padding + iy * stride;
    float sum = 0;
    for (int i = 0; i < k_ny; i++) {
        for (int j = 0; j < k_nx; j++) {
            int img_x = img_x_s + j;
            int img_y = img_y_s + i;
            if (img_x < 0 || img_y < 0 || img_x >= img_nx || img_y >= img_ny) {
                sum += kernel[i * k_nx + j] * padding_value;
                continue;
            }
            sum += kernel[i * k_nx + j] * img[img_x + (img_y) * img_nx];
        }
    }
    out[iy * out_nx + ix] = sum;
}

int main(int argc, char **argv) {
    int mode = 0;
    if (argc >= 2) {
        mode = atoi(argv[1]);
    }
    int nx = 1024, ny = 1024;
    int stride = 1, padding = 1, padding_value=0;
    int kx = 5, ky = 5;
    int out_nx = ((nx + padding * 2) - kx) /stride + 1;
    int out_ny = ((ny + padding * 2) - ky) /stride + 1;

    int block_nx = 32, block_ny = 32;
    // alloc
    float *h_in = (float*)malloc(sizeof(float) * nx * ny);
    float *h_out = (float*)malloc(sizeof(float) * out_nx * out_ny);
    float *h_kernel = (float*)malloc(sizeof(float) * kx * ky));
    float *d_in, *d_out, *d_kernel;
    CHECK(hipMalloc(&d_in, sizeof(float) * nx * ny));
    CHECK(hipMalloc(&d_out, sizeof(float) * out_nx * out_ny));
    CHECK(hipMalloc(&d_kernel, sizeof(float) * kx * ky));
    // init
    dim3 blockDim(block_nx, block_ny);
    dim3 gridDim((out_nx-1)/block_nx+1, (out_ny-1)/block_ny+1);
    for (int y = 0; y < ny; y++) {
        for (int x = 0; x < nx; x++) {
            h_in[y * nx + x] = 1;
        }
    }
    for (int y = 0; y < ky; y++) {
        for (int x = 0; x < kx; x++) {
            h_kernel[y * kx + x] = 1;
        }
    }
    CHECK(hipMemcpy(d_in, h_in, sizeof(float) * nx * ny, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_kernel, h_kernel, sizeof(float) * kx * ky, hipMemcpyHostToDevice));

    // call
    switch (mode) {
        case 0:
            convolve2d<<<gridDim, blockDim>>>(d_in, nx, ny, d_kernel, kx, ky, d_out, out_nx, out_ny, stride, padding, padding_value);
            break;
    }
    // check
    CHECK(hipMemcpy(h_out, d_out, sizeof(float) * out_nx * out_ny, hipMemcpyDeviceToHost));
    bool wrong = false;
    for (int y = 0; y < out_ny; y++) {
        for (int x = 0; x < out_nx; x++) {
            if (x == 0 && y == 0 || x == out_nx-1 && y == out_ny-1 || x == 0 && y == out_ny-1 || x == out_nx-1 && y == 0) {
                if (h_out[y * out_nx + x] != 16) {
                    wrong = true;
                }
            } else if (x == 0 || y == 0 || x == out_nx-1 || y == out_ny-1) {
                if (h_out[y * out_nx + x] != 20) {
                    wrong = true;
                }
            } else {
                if (h_out[y * out_nx + x] != 25) {
                    wrong = true;
                }
            }
        }
    }
    if (wrong) {
        printf("wrong\n");
    }
    // free

}